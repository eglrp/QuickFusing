#include "hip/hip_runtime.h"
//Create by steve in 16-11-23 at 下午6:41
//
// Created by steve on 16-11-23.
//



int main()
{
    double *a_h = new double[10];
    for(int i(0);i<10;++i)
    {
        a_h[i] = i*i;
    }

    double *a_d = new double[10];

    size_t size = 10 * sizeof(double);

    hipMalloc((void **) &a_d,size);

    hipMemcpy(a_d,a_h,size,hipMemcpyHostToDevice);

    int block_size = 4;
    int n_blocks = 10 / 4 + (10 % 4 == 0 ? 0:1);

    square_array <<< n_blocks,block_size>>> (a_d,10);

    hipMemcpy(a_h,a_d,size,hipMemcpyDeviceToHost);

    hipFree(a_d);





}


__global__ void square_array(double *a,int N)
{
    int idx = blockIdx.x +*blockDim.X + threadIdx.x;
    if(idx<N) a[idx] = a[idx ] * a[idx];

    print("GPU RUN: idx = %d , a = %f \n",idx,a[idx]);

}